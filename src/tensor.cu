#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "cuda_tensor.h"
#include "config.h"
#include <hip/hip_fp16.h>

__global__ static void f32_to_f16_kernel(__half* dst, const float* src, size_t n) {
	int threads = gridDim.x * blockDim.x;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	while (index < n) {
		dst[index] = __float2half(src[index]);
		index += threads;
	}
}
bool f32_to_f16(__half* dst, const float* src, size_t n) {
	int g = GPUGridSize();
	int b = GPUBlockSize();
	f32_to_f16_kernel<<<g, b>>>(dst, src, n);
	hipError_t err = hipDeviceSynchronize();
	return err == hipSuccess;
}
__global__ static void f16_to_f32_kernel(float* dst, const __half* src, size_t n) {
	int threads = gridDim.x * blockDim.x;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	while (index < n) {
		dst[index] = __half2float(src[index]);
		index += threads;
	}
}
bool f16_to_f32(float* dst, const __half* src, size_t n) {
	int g = GPUGridSize();
	int b = GPUBlockSize();
	f16_to_f32_kernel<<<g, b>>>(dst, src, n);
	hipError_t err = hipDeviceSynchronize();
	return err == hipSuccess;
}

__global__ void tensor_upsample_kernel(void* dst_mem, int width, int height , void* src_mem,
	int batch, int channels, int stride_w, int stride_h, hipdnnDataType_t data_type, hipdnnTensorFormat_t data_format){
 
	int threads = gridDim.x * blockDim.x;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int c_size_dest = channels * width * height;
	int elements = batch * c_size_dest;

	int src_width = width / stride_w;
	int src_height = height / stride_h;
	int c_size_src = channels * src_width * src_height;

	while (index < elements) {
		int b = index / c_size_dest;
		int temp = index % c_size_dest;
		int c, h, w, index_src;
		if (data_format == HIPDNN_TENSOR_NCHW) {
			c = temp / (width * height);
			temp = temp % (width * height);
			h = temp / width;
			w = temp % width;
			index_src = b * c_size_src + c * (src_width * src_height) +
				(h / stride_h) * src_width + w / stride_w;
		}
		else {
			h = temp / (width * channels);
			temp = temp % (width * channels);
			w = temp / channels;
			c = temp % channels;
			index_src = b * c_size_src + (h / stride_h) * (src_width * channels) +
				(w / stride_w) * channels + c;
		}
		if (data_type == HIPDNN_DATA_FLOAT) {
			float* src = reinterpret_cast<float*>(src_mem);
			float* dst = reinterpret_cast<float*>(dst_mem);
			dst[index] = src[index_src];
		}
		else {
			__half* src = reinterpret_cast<__half*>(src_mem);
			__half* dst = reinterpret_cast<__half*>(dst_mem);
			dst[index] = src[index_src];
		}
		index += threads;
	} 
}
__global__ void tensor_downsample_kernel(void* dst_mem, int width, int height, void* src_mem,
	int batch, int channels, int stride_w, int stride_h, hipdnnDataType_t data_type, hipdnnTensorFormat_t data_format) {

	int threads = gridDim.x * blockDim.x;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int c_size_dest = channels * width * height;
	int elements = batch * c_size_dest;

	int src_width = width * stride_w;
	int src_height = height * stride_h;
	int c_size_src = channels * src_width * src_height;

	while (index < elements) {
		int b = index / c_size_dest;
		int temp = index % c_size_dest;
		int c, h, w,  src_h, src_w, index_src;
		if (data_format == HIPDNN_TENSOR_NCHW) {
			c = temp / (width * height);
			temp = temp % (width * height);
			h = temp / width;
			w = temp % width; 
		}
		else {
			h = temp / (width * channels);
			temp = temp % (width * channels);
			w = temp / channels;
			c = temp % channels; 
		}
		src_h = h * stride_h;
		
		
		if (data_type == HIPDNN_DATA_FLOAT) {
			float* src = reinterpret_cast<float*>(src_mem);
			float* dst = reinterpret_cast<float*>(dst_mem);
			dst[index] = 0.0f; 
			for (int i = 0; i < stride_h; i++, src_h++) {
				src_w = w * stride_w;
				for (int j = 0; j < stride_w; j++, src_w++) { 
					if (data_format == HIPDNN_TENSOR_NCHW) {
						index_src = b * c_size_src + c * (src_width * src_height) +
							src_h * src_width + src_w;
					}
					else {
						index_src = b * c_size_src + src_h * (src_width * channels) +
							src_w * channels + c;
					}
					dst[index] += src[index_src];
				}
			}
		}
		else {
			__half* src = reinterpret_cast<__half*>(src_mem);
			__half* dst = reinterpret_cast<__half*>(dst_mem);
			dst[index] = __float2half(0.0f);
			for (int i = 0; i < stride_h; i++, src_h++) {
				src_w = w * stride_w;
				for (int j = 0; j < stride_w; j++, src_w++) {
					if (data_format == HIPDNN_TENSOR_NCHW) {
						index_src = b * c_size_src + c * (src_width * src_height) +
							src_h * src_width + src_w;
					}
					else {
						index_src = b * c_size_src + src_h * (src_width * channels) +
							src_w * channels + c;
					} 
					dst[index] = __hadd(dst[index], src[index_src]);
				}
			}
		}
		index += threads;
	}
}

bool CudaTensor::UpSample(CudaTensor& output, int stride_w, int stride_h) {
	if (stride_w <= 0 || stride_w <= 0 || 0 == elements) return false;

	int w_o = w * stride_w;
	int h_o = h * stride_h;

	if (output.n != n || output.c != c || output.w != w_o || output.h != h_o) {
		cerr << " Error: Wrong result demension in tensor upsample !\n";
		return false;
	}
	if (output.data_type != data_type || output.data_format != data_format) {
		cerr << " Error: Inconsistent data types in tensor upsample !\n";
		return false;
	}
	int g = GPUGridSize();
	int b = GPUBlockSize();
	switch (data_type) {
	case HIPDNN_DATA_FLOAT:
	case HIPDNN_DATA_HALF:
		tensor_upsample_kernel<<<g,b>>>(output.gpu_data, output.w, output.h, gpu_data, n, c, stride_w, stride_h, data_type, data_format);
		break;
	default:
		cerr << " Error: Only support FP16 or FP32!\n";
		return false;
	}  
	hipError_t err = hipDeviceSynchronize();

	if (err != hipSuccess) {
		cerr << " Error: CudaTensor.UpSample failed - cudaSynchronize failed err " << err << "!\n";
		return false;
	}
	return true;
}
bool CudaTensor::DownSample(CudaTensor& output, int stride_w, int stride_h) {
	if (stride_w <= 0 || stride_w <= 0 || 0 == elements) return false;

	int w_o = w / stride_w;
	int h_o = h / stride_h;

	if (output.n != n || output.c != c || output.w != w_o || output.h != h_o) {
		cerr << " Error: Wrong result demension in tensor upsample !\n";
		return false;
	}
	if (output.data_type != data_type || output.data_format != data_format) {
		cerr << " Error: Inconsistent data types in tensor upsample !\n";
		return false;
	}
	int g = GPUGridSize();
	int b = GPUBlockSize(); 

	switch (data_type) {
	case HIPDNN_DATA_FLOAT:
	case HIPDNN_DATA_HALF:
		tensor_downsample_kernel <<<g, b>>>(output.gpu_data, output.w, output.h, gpu_data, n, c, stride_w, stride_h, data_type, data_format);
		break;
	default:
		cerr << " Error: Only support FP16 or FP32!\n";
		return false;
	}
	hipError_t err = hipDeviceSynchronize();

	if (err != hipSuccess) {
		cerr << " Error: CudaTensor.DownSample failed - cudaSynchronize failed err " << err << "!\n";
		return false;
	}
	return true;
}
__global__ static void tensor_add_kernel(void* data, const void* op, int elements, hipdnnDataType_t data_type) {
	int index = blockDim.x  * blockIdx.x + threadIdx.x;
	int threads = gridDim.x * blockDim.x;
	while (index < elements) {
		if (data_type == HIPDNN_DATA_FLOAT) {
			float* dst = reinterpret_cast<float *>(data);
			const float* src = reinterpret_cast<const float *>(op);
			dst[index] += src[index];
		}
		else {
			__half* dst = reinterpret_cast<__half *>(data);
			const __half* src = reinterpret_cast<const __half *>(op);
			dst[index] = __hadd(dst[index], src[index]);
		}
		index += threads;
	}
}
__global__ static void tensor_add_kernel_ex(void* data, const void* op, int batch, int channels, int height, int width, int op_batch, hipdnnDataType_t data_type, hipdnnTensorFormat_t data_format) {
	int index = blockDim.x  * blockIdx.x + threadIdx.x;
	int threads = gridDim.x * blockDim.x;
	int c_size = channels * height * width;
	int elements = batch * c_size;
	while (index < elements) {
		int b = index / c_size;
		int temp = index % c_size;
		int c ;
		if (data_format == HIPDNN_TENSOR_NCHW) {
			c = temp / (width * height);
			temp = temp % (width * height); 
		}
		else {
			//int h = temp / (width * channels);
			temp = temp % (width * channels); 
			c = temp % channels;
		}
		if (data_type == HIPDNN_DATA_FLOAT) {
			float* dst = reinterpret_cast<float *>(data);
			const float* src = reinterpret_cast<const float *>(op);
			if(batch == op_batch )
				dst[index] += src[b * channels + c];
			else 
				dst[index] += src[c];
		}
		else {
			__half* dst = reinterpret_cast<__half *>(data);
			const __half* src = reinterpret_cast<const __half *>(op);
			if (batch == op_batch)
				dst[index] = __hadd(dst[index], src[b * channels + c]);
			else
				dst[index] = __hadd(dst[index], src[c]);
		}
		index += threads;
	}
}

bool CudaTensor::Add(const CudaTensor& op) {
	if (!op.gpu_data || !op.elements) {
		return true;
	}
	if (!gpu_data) {
		(*this) = op;
		return SameShape(op);
	}
	if (op.data_type != data_type || op.data_format != data_format) {
		cerr << " Error: Inconsistent data types in tensor add !\n";
		return false;
	}
	if(data_type != HIPDNN_DATA_FLOAT && data_type != HIPDNN_DATA_HALF){
		cerr << " Error: Unsportted data format in tensor add !\n";
		return false;
	}
	if (n != op.n && op.n != 1) {
		cerr << " Error: Inconsistent batches in tensor add !\n";
		return false;
	}
	if (elements == op.elements) {
		int g = GPUGridSize();
		int b = GPUBlockSize(); 
		tensor_add_kernel <<<g, b>>> (gpu_data, op.gpu_data, elements, data_type);
	}
	else if (c == op.c && (op.h == 1 && op.w == 1)) {
		int g = GPUGridSize();
		int b = GPUBlockSize();
		tensor_add_kernel_ex <<<g,b >>> (gpu_data, op.gpu_data, n, c, h, w, op.n, data_type, data_format);
	}
	else {
		cerr << "Not compatible!\n";
		return false;
	}
	hipError_t err = hipDeviceSynchronize();
	if (hipSuccess != err) {
		cerr << "Error: FloatTensor4D.Add returned " << err << endl;
		return false;
	}
	return true;
}
__global__ static void tensor_add_kernel(void* data, float op, int elements, hipdnnDataType_t data_type) {
	int index = blockDim.x  * blockIdx.x + threadIdx.x;
	int threads = gridDim.x * blockDim.x;

	while (index < elements) {
		if (data_type == HIPDNN_DATA_FLOAT) {
			float* dst = reinterpret_cast<float *>(data); 
			dst[index] += op;
		}
		else {
			__half* dst = reinterpret_cast<__half *>(data);
			__half hop = __float2half(op);
			dst[index] = __hadd(dst[index], hop);
		}
		index += threads;
	}
}
bool CudaTensor::Add(float op) {	 
	if (!gpu_data) {
		(*this) = op;
		return true;
	}	 
	if (data_type != HIPDNN_DATA_FLOAT && data_type != HIPDNN_DATA_HALF) {
		cerr << " Error: Unsportted data format in tensor add !\n";
		return false;
	} 
	int g = GPUGridSize();
	int b = GPUBlockSize(); 
	tensor_add_kernel <<<g, b>>> (gpu_data, op, elements, data_type);
	hipError_t err = hipDeviceSynchronize();
	if (hipSuccess != err) {
		cerr << "Error: FloatTensor4D.Add returned " << err << endl;
		return false;
	}
	return true;
}

__global__ static void tensor_muladd_kernel(void* data, float op_m, float op_a, int elements, hipdnnDataType_t data_type) {
	int index = blockDim.x  * blockIdx.x + threadIdx.x;
	int threads = gridDim.x * blockDim.x;

	while (index < elements) {
		if (data_type == HIPDNN_DATA_FLOAT) {
			float* dst = reinterpret_cast<float *>(data);
			dst[index] = op_m * dst[index] + op_a;
		}
		else {
			__half* dst = reinterpret_cast<__half *>(data);
			__half hop = __float2half(op_m);
			dst[index] = __hmul(dst[index], hop);
			if (op_a != 0.0f) {
				hop = __float2half(op_a);
				dst[index] = __hadd(dst[index], hop);
			}
		}
		index += threads;
	}
}
bool CudaTensor::MulAdd(float op_m, float op_a) {
	if (!gpu_data) {
		return false;
	}
	if (data_type != HIPDNN_DATA_FLOAT && data_type != HIPDNN_DATA_HALF) {
		cerr << " Error: Unsportted data format in tensor add !\n";
		return false;
	}
	int g = GPUGridSize();
	int b = GPUBlockSize();
	tensor_muladd_kernel <<<g, b >>> (gpu_data, op_m, op_a, elements, data_type);
	hipError_t err = hipDeviceSynchronize();
	if (hipSuccess != err) {
		cerr << "Error: FloatTensor4D.Add returned " << err << endl;
		return false;
	}
	return true;
}

__global__ static void tensor_muladd_kernel_ex(void* data, const void* op_m, const void* op_a, int batch, int channels, int height, int width, hipdnnDataType_t data_type, hipdnnTensorFormat_t data_format) {
	int index = blockDim.x  * blockIdx.x + threadIdx.x;
	int threads = gridDim.x * blockDim.x;
	int c_size = channels * height * width;
	int elements = batch * c_size;
	while (index < elements) {
		int b = index / c_size;
		int temp = index % c_size;
		int c;
		if (data_format == HIPDNN_TENSOR_NCHW) {
			c = temp / (width * height);
			temp = temp % (width * height);
		}
		else {
			//int h = temp / (width * channels);
			temp = temp % (width * channels);
			c = temp % channels;
		}
		if (data_type == HIPDNN_DATA_FLOAT) {
			float* dst = reinterpret_cast<float *>(data);
			const float* src_m = reinterpret_cast<const float *>(op_m);
			const float* src_a = reinterpret_cast<const float *>(op_a);
			dst[index] = dst[index] * src_m[c] + src_a[c];
		}
		else {
			__half* dst = reinterpret_cast<__half *>(data);
			const __half* src_m = reinterpret_cast<const __half *>(op_m);
			const __half* src_a = reinterpret_cast<const __half *>(op_a);
			__half temp = __hmul(dst[index], src_m[c]);
			dst[index] = __hadd(temp, src_a[c]);
		}
		index += threads;
	}
}
bool CudaTensor::MulAdd(const CudaTensor& op_m, const CudaTensor& op_a) {
	if (!op_m.gpu_data || !op_m.elements || !op_a.gpu_data || !op_a.elements) {
		return true;
	}
	if (!gpu_data) {
		return false;
	}
	if (op_m.data_type != data_type || op_m.data_format != data_format
		||op_a.data_type != data_type || op_a.data_format != data_format) {
		cerr << " Error: Inconsistent data types in tensor multia !\n";
		return false;
	}
	if (data_type != HIPDNN_DATA_FLOAT && data_type != HIPDNN_DATA_HALF) {
		cerr << " Error: Unsportted data format in tensor add !\n";
		return false;
	}
	if (op_m.n != 1 || op_m.c != c || op_m.w != 1 || op_m.h != 1 ||
		op_a.n != 1 || op_a.c != c || op_a.w != 1 || op_a.h != 1) {
		cerr << " Error: Dims of operators must be [1x"<<c<<"x1x1]!\n";
		return false;
	}
	int g = GPUGridSize();
	int b = GPUBlockSize();
	tensor_muladd_kernel_ex <<<g, b >>> (gpu_data, op_m.gpu_data, op_a.gpu_data, n, c, h, w,  data_type, data_format);
 
	hipError_t err = hipDeviceSynchronize();
	if (hipSuccess != err) {
		cerr << "Error: FloatTensor4D.Add returned " << err << endl;
		return false;
	}
	return true;
}

__global__ static void adam_update_kernel(void* params, void* g, void* m, void* v, int elements, 
	hipdnnDataType_t data_type, int t, AdamConfig adam_config, float decay) {
	int i = blockDim.x  * blockIdx.x + threadIdx.x;
	int threads = gridDim.x * blockDim.x;
	if (data_type == HIPDNN_DATA_FLOAT) {
		float* theta = reinterpret_cast<float*>(params);
		float* gt = reinterpret_cast<float*>(g);
		float* mt = reinterpret_cast<float*>(m);
		float* vt = reinterpret_cast<float*>(v);
		float m_hat, v_hat;
		while (i < elements) {
			if (decay != 0.0f) gt[i] -= decay * theta[i];
			mt[i] = adam_config.beta1 * mt[i] + (1.0f - adam_config.beta1) * gt[i];
			vt[i] = adam_config.beta2 * vt[i] + (1.0f - adam_config.beta2) * gt[i] * gt[i];
			m_hat = mt[i] / (1.0f - powf(adam_config.beta1, t));
			v_hat = vt[i] / (1.0f - powf(adam_config.beta2, t));
			theta[i] = theta[i] + adam_config.alpha * m_hat / (sqrtf(v_hat) + adam_config.epsilon);
			gt[i] = 0.0f;
			i += threads;

		}
	}
	else {
		__half* theta = reinterpret_cast<__half*>(params);
		__half* gt = reinterpret_cast<__half*>(g);
		__half* mt = reinterpret_cast<__half*>(m);
		__half* vt = reinterpret_cast<__half*>(v);
		float m_hat, v_hat, mt_i, gt_i, vt_i, theta_i;
		while (i < elements) {
			mt_i = __half2float(mt[i]);
			gt_i = __half2float(gt[i]);
			vt_i = __half2float(vt[i]);
			theta_i = __half2float(theta[i]);
			if (decay != 0.0f) {
				gt_i -= decay * theta_i;
			}
			mt_i = adam_config.beta1 * mt_i + (1.0f - adam_config.beta1) * gt_i;
			vt_i = adam_config.beta2 * vt_i + (1.0f - adam_config.beta2) * gt_i * gt_i;

			m_hat = mt_i / (1.0f - powf(adam_config.beta1, t));
			v_hat = vt_i / (1.0f - powf(adam_config.beta2, t));

			mt[i] = __float2half(mt_i);
			vt[i] = __float2half(vt_i); 

			theta_i = theta_i - adam_config.alpha * m_hat / (sqrtf(v_hat) + adam_config.epsilon);
			theta[i] = __float2half(theta_i);
			gt[i] = __float2half(0.0f);
			i += threads;

		}
	}
}
bool adam_update(void* params, void* gt, void* mt, void* vt, int elements, int t, hipdnnDataType_t data_type ,float lr , bool decay) {

	AdamConfig adam_config = GetAppConfig().GetAdamConfig(); 
	int g = GPUGridSize();
	int b = GPUBlockSize();
	t++;
	adam_config.alpha /= GetAppConfig().GetBatch();
	float fdecay = 0.0f;
	if (decay)
		fdecay = GetAppConfig().Decay();
	adam_update_kernel<<<g, b >>>(params, gt, mt, vt, elements, data_type, t, adam_config, fdecay);

	hipError_t err = hipDeviceSynchronize();
	if (hipSuccess != err) {
		cerr << " Error: adam_update returned " << err << endl;
		return false;
	}
	return true; 
}

__global__ static void sgd_update_kernel(void* params, void* updates, int elements, hipdnnDataType_t data_type, float lr, float decay, float momentum) {
	int index = blockDim.x  * blockIdx.x + threadIdx.x;
	int threads = gridDim.x * blockDim.x;
	while (index < elements) {
		if (data_type == HIPDNN_DATA_FLOAT) {
			float* dst = reinterpret_cast<float*>(params);
			float* src = reinterpret_cast<float*>(updates);
			if (decay != 0.0f) src[index] -= (dst[index] * decay);
			dst[index] += (lr * src[index]);
			src[index] *= momentum;
		}
		else {
			__half* dst = reinterpret_cast<__half*>(params);
			__half* src = reinterpret_cast<__half*>(updates);
			__half temp;
			if (decay != 0.0f) {
				__hmul(dst[index], __float2half(decay));
				src[index] = __hsub(src[index], temp);
			}
			temp = __hmul(src[index], __float2half(lr));
			dst[index] = __hsub(dst[index], temp);
			src[index] = __hmul(src[index], __float2half(momentum));
		}
		index += threads;
	}
}


bool sgd_update(void* params, void* updates, int elements, hipdnnDataType_t data_type, float lr, bool decay) {
	int g = GPUGridSize();
	int b = GPUBlockSize();
	const SgdConfig& sgd_config = GetAppConfig().GetSgdConfig();
	float fdecay = 0.0f;
	int batch = GetAppConfig().GetBatch();
	if (decay) {
		fdecay = GetAppConfig().Decay();
	}
	sgd_update_kernel<<<g, b >>>(params, updates, elements, data_type, lr / batch, fdecay, sgd_config.momentum);

	hipError_t err = hipDeviceSynchronize();
	if (hipSuccess != err) {
		cerr << " Error: sgd_update returned " << err << endl;
		return false;
	}
	return true;
}
/*
void* beta = params.BatchData(0);
void* gamma = params.BatchData(1);
void* running_mu = params.BatchData(2);
void* running_var = params.BatchData(3);
*/
__global__ static void fuse_batchnorm_kernel(void* filters, void* bias, void* batchnorm_params, int output_channels, int filter_size, hipdnnDataType_t data_type) {
	int index = blockIdx.x * blockDim.x + threadIdx.x; 
 
	while (index < output_channels) {
		if (data_type == HIPDNN_DATA_FLOAT) {
			float* filters_f = reinterpret_cast<float*>(filters);
			float* bias_f = reinterpret_cast<float*>(bias);
			float* beta = reinterpret_cast<float*>(batchnorm_params);
			float* gamma = beta + output_channels;
			float* mu = gamma + output_channels;
			float* var = mu + output_channels;
			float temp = gamma[index] / sqrt(var[index] + 1.0e-5);
			//float alpha = temp / size;
			bias_f[index] += (beta[index] - mu[index] * temp); 
			int f_i = index * filter_size;
			for (int i = 0; i < filter_size; i++, f_i++) {
				filters_f[f_i] *= temp; 
			} 
		}
		else {
			__half* filters_h = reinterpret_cast<__half*>(filters);
			__half* bias_h = reinterpret_cast<__half*>(bias);
			__half* beta = reinterpret_cast<__half*>(batchnorm_params);
			__half* gamma = beta + output_channels;
			__half* mu = gamma + output_channels;
			__half* var = mu + output_channels;

			__half temp = __hdiv(gamma[index], hsqrt( __hadd(var[index] , __float2half(1.0e-5))));
		 
			bias_h[index] = __hadd(bias_h[index],(beta[index] - mu[index] * temp));
			int f_i = index * filter_size;
			for (int i = 0; i < filter_size; i++, f_i++) {
				filters_h[f_i] = __hmul(filters_h[f_i], temp); 
			}
		}

		index += blockDim.x * gridDim.x;
	}


}
bool fuse_batchnorm(void* filters, void* bias, void* batchnorm_params, int output_channels, int filter_size, hipdnnDataType_t data_type) {
	int b = GPUBlockSize();
	int g = output_channels / b;
	if (output_channels % b) g++;
	fuse_batchnorm_kernel<<<g, b >>>(filters, bias, batchnorm_params, output_channels, filter_size, data_type);

	hipError_t err = hipDeviceSynchronize();
	if (hipSuccess != err) {
		cerr << " Error: fuse_batchnorm returned " << err << endl;
		return false;
	}
	return true;
}
__global__ static void one_stride_pooling_patch_kernel(void* out, void* in, int batch, int channels, int width, int height, hipdnnDataType_t data_type, hipdnnTensorFormat_t data_format, bool forwarding) {
	int index = blockIdx.x * blockDim.x + threadIdx.x; 
	int threads = gridDim.x * blockDim.x;
	int size = height * width;
	int c_size = channels * size;
	int elements = batch * c_size;
	while (index < elements) {
		int b = index / c_size;
		int temp = index % c_size;
		int c, w, h, index1;
		if (data_format == HIPDNN_TENSOR_NCHW) {
			c = temp / size;
			temp = temp % size;
			h = temp / width;
			w = temp % width; 
			//index1 = b * c_size + c * size + (h + 1) * (width + 1) + width + 1;
			index1 = b * c_size + c * size + h * (width + 1) + width;
		}
		else {
			size = width * channels;
			h = temp / size;
			temp = temp % size;
			w = temp / channels;
			c = temp % channels;
			//index1 = b * c_size +  (h + 1) * (width + 1) * channels + (width + 1) * channels + c;
			index1 = b * c_size + h * (width + 1) * channels + width * channels + c;
		}
		if (data_type == HIPDNN_DATA_FLOAT) {
			float* fout = reinterpret_cast<float*>(out);
			float* fin = reinterpret_cast<float*>(in);
			if(forwarding)
				fout[index] = fin[index1];
			else
				fout[index1] = fin[index];
		}
		else {
			__half* hout = reinterpret_cast<__half*>(out);
			__half* hin = reinterpret_cast<__half*>(in);
			if (forwarding)
				hout[index] = hin[index1];
			else
				hout[index1] = hin[index];
		}
		index += threads;
	}
}
bool one_stride_pooling_patch(CudaTensor& out, const CudaTensor& in, bool forwarding) {
	int g = GPUGridSize();
	int b = GPUBlockSize();
	int h, w;
	if (forwarding) {
		h = out.Height();
		w = out.Width();
	}
	else {
		h = in.Height();
		w = in.Width();
	}
	one_stride_pooling_patch_kernel<<<g, b >>>(out, in,out.Batch(),out.Channel(),w,h ,
		out.DataType(),out.DataFormat(), forwarding);

	hipError_t err = hipDeviceSynchronize();
	if (hipSuccess != err) {
		cerr << " Error: one_stride_pooling_patch returned " << err << endl;
		return false;
	}
	return true;
}
